#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <exception>

#include <Windows.h>

#include "ArgumentParser.h"
#include "Matrix.h"

#define THREAD_BLOCK_SIZE 3

void usage(void) {
	std::cout << "Usage: ./MatrixMultiplicationCuda [FILE] [FILE]" << std::endl;
}

typedef struct {
	int* elements;
	int width;
	int height;
} MatrixStruct;

__global__ void multiplyMatrixesGPU(MatrixStruct a, MatrixStruct b, MatrixStruct c) {
	int calc = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row > a.height) || (col > b.width)) return;

	for (int i = 0; i < a.width; i++) {
		calc += (a.elements[row * a.width + i]) * (b.elements[i * b.width + col]);
	}
	c.elements[row * c.width + col] = calc;
}

void print3x3(const Matrix* a, const Matrix* b, const Matrix* c) {
	for (int i = 0; i < c->getWidth(); i++) {

		std::cout << "|";
		for (int j = 0; j < c->getHeight(); j++) {
			std::cout << a->getElements()[i*c->getWidth() + j] << (j == c->getHeight() - 1 ? "" : " ");
		}
		std::cout << "|";


		std::cout << (i == c->getWidth()/2 ? " * " : "   ");

		std::cout << "|";
		for (int j = 0; j < c->getHeight(); j++) {
			std::cout << b->getElements()[i*c->getWidth() + j] << (j == c->getHeight() - 1 ? "" : " ");
		}
		std::cout << "|";

		std::cout << (i == c->getWidth() / 2 ? " = " : "   ");
		
		std::cout << "|";
		for (int j = 0; j < c->getHeight(); j++) {
			std::cout << c->getElements()[i*c->getWidth() + j] << (j == c->getHeight() - 1 ? "" : " ");
		}
		std::cout << "|";

		std::cout << std::endl;
	}
}

void multiplyMatrixes(const Matrix* a, const Matrix* b, Matrix* c) {
	MatrixStruct gpu_a = { nullptr, a->getWidth(), a->getHeight() };
	MatrixStruct gpu_b = { nullptr, b->getWidth(), b->getHeight() };
	MatrixStruct gpu_c = { nullptr, c->getWidth(), c->getHeight() };

	std::cout << "CUDA PREPARATION" << std::endl;
	
	std::cout << "-------------------------------------------------------------" << std::endl;

	hipError_t error = hipMalloc(&gpu_a.elements, gpu_a.height * gpu_a.width * sizeof(int));
	std::cerr << "CUDA MALLOC A: " << hipGetErrorString(error) << std::endl;
	error = hipMemcpy(gpu_a.elements, a->getElements(), a->getWidth() * a->getHeight() * sizeof(int), hipMemcpyHostToDevice);
	std::cerr << "CUDA MEMCPY A: " << hipGetErrorString(error) << std::endl;

	error = hipMalloc(&gpu_b.elements, gpu_b.height * gpu_b.width * sizeof(int));
	std::cerr << "CUDA MALLOC B: " << hipGetErrorString(error) << std::endl;
	error = hipMemcpy(gpu_b.elements, b->getElements(), b->getWidth() * b->getHeight() * sizeof(int), hipMemcpyHostToDevice);
	std::cerr << "CUDA MEMCPY B: " << hipGetErrorString(error) << std::endl;

	error = hipMalloc(&gpu_c.elements, gpu_c.height * gpu_c.width * sizeof(int));
	std::cerr << "CUDA MALLOC C: " << hipGetErrorString(error) << std::endl;

	std::cout << "-------------------------------------------------------------\n" << std::endl;

	dim3 dimBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	dim3 dimGrid((b->getWidth() + dimBlock.x - 1) / dimBlock.x, (a->getHeight() + dimBlock.y - 1) / dimBlock.y);

	multiplyMatrixesGPU <<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);

	std::cout << "KERNEL RUN" << std::endl;
	std::cout << "-------------------------------------------------------------" << std::endl;

	error = hipDeviceSynchronize();	
	std::cerr << "RUN KERNEL: " << hipGetErrorString(error) << std::endl;

	error = hipMemcpy(c->getElements(), gpu_c.elements, c->getWidth() * c->getHeight() * sizeof(int), hipMemcpyDeviceToHost);
	std::cerr << "FROM DEVICE TO HOST: " << hipGetErrorString(error) << std::endl;

	std::cout << "-------------------------------------------------------------\n" << std::endl;
	// Free device memory
	hipFree(gpu_a.elements);
	hipFree(gpu_b.elements);
	hipFree(gpu_c.elements);
}

int main(int argc, char** argv) {
	ArgumentParser* argParser = nullptr;
	Matrix* firstMatrix		  = nullptr;
	Matrix* secondMatrix	  = nullptr;
	Matrix* resultMatrix	  = nullptr;

	try {
		argParser = new ArgumentParser(argc, argv);
	} catch (std::exception& e) {
		std::cerr << "What(): " << e.what() << std::endl;
	}

	firstMatrix = argParser->prepareMatrix(MATRIX::FIRST);
	secondMatrix = argParser->prepareMatrix(MATRIX::SECOND);
	resultMatrix = new Matrix(firstMatrix->getHeight(), secondMatrix->getWidth());

	multiplyMatrixes(firstMatrix, secondMatrix, resultMatrix);

	print3x3(firstMatrix, secondMatrix, resultMatrix);

	system("pause");

	if (argParser != nullptr) {
		delete argParser;
		argParser = nullptr;
	}

	if (firstMatrix != nullptr) {
		delete firstMatrix;
		firstMatrix = nullptr;
	}

	if (secondMatrix != nullptr) {
		delete secondMatrix;
		secondMatrix = nullptr;
	}

	if (resultMatrix != nullptr) {
		delete resultMatrix;
		resultMatrix = nullptr;
	}
	return 0;
}